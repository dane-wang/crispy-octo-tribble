#include "hip/hip_runtime.h"

#include <time.h>  
#include <vector>
#include "graph_search/planner1.h"
#include "graph_search/parallel_dijkstra.cuh"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>


// __device__ bool path_found_gpu;
__device__ int neighbor_gpu[8];
__device__ int g_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



// template <typename T, typename T1> 
// __global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
// {

//   int tid = blockIdx.x *blockDim.x + threadIdx.x;
//   if (tid < q_size){
//     int node = q[tid];

//     h[tid] = graph[node].f;

//     // printf("%d", q[tid]);
//   }

// }

// template <typename T>
// __global__ void explore(T* q,  planner::Node* graph, T* new_q )
// {
//   int tid = threadIdx.x;
//   int explored_index = q[tid];
//   int n = neighbor_gpu[3];


//   graph[explored_index].explored = true;
//   if (graph[explored_index].h > g_gpu){
//     graph[explored_index].h = g_gpu;
//   }

//   for (int i=0; i<8; i++)
//   {   
    
//     int new_index = explored_index + neighbor_gpu[i];

//     bool edge_detect = true;

    
//     if ((explored_index%n ==0 && (neighbor_gpu[i] == -1 || neighbor_gpu[i] == n-1 || neighbor_gpu[i] == -n-1 )) || ((explored_index+1)%n ==0 && (neighbor_gpu[i] == 1 || neighbor_gpu[i] == n+1 || neighbor_gpu[i] == -n+1 )) || new_index<0 || new_index >= n*n){
//       edge_detect = false;
//     }


//     if (graph[new_index].obstacle == false && edge_detect && !graph[new_index].explored )
//     {      
//       new_q[8*tid+i] = new_index;
//     }
   
//   }

  

// }

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q  )
{
  int tid = threadIdx.x;
  int explored_index = q[tid];
  int n = neighbor_gpu[2];

  // graph[explored_index].explored = true;
  // graph[explored_index].frontier = false;

  // if (explored_index == 0){
  //   printf("I am working on 0" );
  // }




  for (int i=0; i<8; i++)
  {   
    
    int new_index = explored_index + neighbor_gpu[i];

    bool edge_detect = true;

    
              
    // if ((explored_index%n ==0 && neighbor_gpu[i] == -1) || (explored_index%(n-1) ==0 && neighbor_gpu[i] == 1 &&explored_index!=0) || new_index<0 || new_index >= n*n){
    //   edge_detect = false;
    // }

    if ((explored_index%n ==0 && (neighbor_gpu[i] == -1 || neighbor_gpu[i] == n-1 || neighbor_gpu[i] == -n-1 )) || ((explored_index+1)%n ==0 && (neighbor_gpu[i] == 1 || neighbor_gpu[i] == n+1 || neighbor_gpu[i] == -n+1 )) || new_index<0 || new_index >= n*n){
      edge_detect = false;
    }


    if (graph[new_index].obstacle == false && graph[new_index].h == INFINITY && edge_detect)
    {
      graph[new_index].h = g_gpu;

      // graph[new_index].parent = explored_index;
      // graph[new_index].frontier = true;
      
      new_q[8*tid+i] = new_index;
    }
    else if (edge_detect && (graph[new_index].h != INFINITY))
    {
      if (graph[new_index].h > g_gpu)
      {
        graph[new_index].h = g_gpu;
        
        graph[new_index].parent = explored_index;
      }
    }
  }

  
}



extern "C"
void parallel_dijkstra(planner::Node* graph, int n, int goal_index, int max_thread){


  // Use dijkstra to initialize the heuristic values for all the node.


  //The heuristic for goal is 0
  // graph[goal_index].h = 0;
  graph[goal_index].g = 0;
  graph[goal_index].h = 0;
  graph[goal_index].f = graph[goal_index].g + graph[goal_index].h;
  int g = 0;
  
  // bool path_found = false;
  // int goal = goal_index;

  thrust::host_vector<int> q_lists;
  q_lists.push_back(goal_index);

  const int map_size = n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  int neighbor[8] = {1, -1, n, -n, n+1, n-1, -n+1, -n-1};

  //Copy all needed variables to gpu
  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, graph, map_size, hipMemcpyHostToDevice);

  // hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  8*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(g_gpu), &g,  sizeof(int));

  //q list on gpu
  thrust::device_vector<int> q_lists_gpu = q_lists;

  while(q_lists_gpu.size()!=0){


    int q_size = q_lists_gpu.size();

    //new_q is the list store the frontier generated from this step of exploration
    thrust::device_vector<int> new_q_lists_gpu(8*q_size);
    thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);

    g = g+5;
    hipMemcpyToSymbol(HIP_SYMBOL(g_gpu), &g,  sizeof(int));
  


    //Determine how many thread should be launched
    int thread_size = min(max_thread, q_size);

    

    //Launch the kernel to explore the map
    explore<<<1,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()));
    hipDeviceSynchronize();
    

    

    // // Remove all element that is not used during the exploration and repeated value

    
    
    new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
    
    new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );

    // std::cout<< new_q_lists_gpu.size() << std::endl;

    q_lists_gpu.clear();
 
    q_lists_gpu = new_q_lists_gpu;
    new_q_lists_gpu.clear();

    
    // if (new_q_lists_gpu.empty()) break;
    
    // // Create new q list based on origional and updated q
    // if (q_size <= max_thread) {
    //   q_lists_gpu.clear();
 
    //   q_lists_gpu = new_q_lists_gpu;
    //   new_q_lists_gpu.clear();
    //   }
    // else {
      
    //   q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+max_thread );
    //   q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
    //   new_q_lists_gpu.clear();

      
    //   }

    
    //q_size = q_lists_gpu.size();
    // thrust::device_vector<float> h_value(q_size);

    // if (q_size > 1024) {
    //   int block = q_size / 1024 + 1;
      
    //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

    //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

    }

  hipMemcpy(graph, map_gpu,  map_size, hipMemcpyDeviceToHost );

    


 
}