#include "hip/hip_runtime.h"


#include <time.h>  
#include <vector>
#include "graph_search/planner1.h"
#include "graph_search/parallel_explore.cuh"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>


__device__ bool path_found_gpu;
__device__ int neighbor_gpu[4];
__device__ int goal_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q  )
{
  int tid = threadIdx.x;
  int explored_index = q[tid];
  int n = neighbor_gpu[3];

  graph[explored_index].explored = true;
  graph[explored_index].frontier = false;

  if (graph[explored_index].goal){
    printf("FOUND");
    printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
    // planner::Node* temp_node = graph[explored_index].parent;
    // while (!temp_node->start){
       
    //     temp_node->path = true;
    //     temp_node = temp_node->parent;
    // }
    path_found_gpu = true;
  }

  if (!path_found_gpu){
    for (int i=0; i<4; i++)
    {   
      
      int new_index = explored_index + neighbor_gpu[i];
      bool edge_detect = true;

      
                
      if ((explored_index%n ==0 && neighbor_gpu[i] == -1) || (explored_index%(n-1) ==0 && neighbor_gpu[i] == 1 &&explored_index!=0) || new_index<0 || new_index >= n*n){
        edge_detect = false;
      }


      if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
      {
        graph[new_index].g = graph[explored_index].g + 1;
          
        float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2));
          // printf("%f", h_1);
        graph[new_index].h = h_1;

          
        graph[new_index].f = graph[new_index].h + graph[new_index].g;
        graph[new_index].parent = explored_index;
        graph[new_index].frontier = true;
        
        new_q[4*tid+i] = new_index;
      }
      else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
      {
        if (graph[new_index].g > graph[explored_index].g + 1)
        {
          graph[new_index].g = graph[explored_index].g + 1;
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
        }
      }
    }

  }

}
extern "C"
void parallel_explore(planner::Node* graph, int n, bool path_found, int start_index, int max_thread){

    
    thrust::host_vector<int> q_lists;
    q_lists.push_back(start_index);

    const int map_size = n*n*sizeof(planner::Node);

    planner::Node *map_gpu;

    int neighbor[4] = {1, -1, n, -n};

    hipMalloc( (void**)&map_gpu, map_size );
    hipMemcpy(map_gpu, &graph, map_size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  4*sizeof(int));
    // hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));


    thrust::device_vector<int> q_lists_gpu = q_lists;
    thrust::device_vector<float> f_value(q_lists_gpu.size());
    get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
    

}
