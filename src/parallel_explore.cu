#include "hip/hip_runtime.h"


#include <time.h>  
#include <vector>
#include "graph_search/planner1.h"
#include "graph_search/parallel_explore.cuh"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>


__device__ bool path_found_gpu;
__device__ int neighbor_gpu[4];
__device__ int goal_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q  )
{
  int tid = threadIdx.x;
  int explored_index = q[tid];
  int n = neighbor_gpu[3];

  graph[explored_index].explored = true;
  graph[explored_index].frontier = false;

  if (graph[explored_index].goal){
    printf("FOUND");
    printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
    // planner::Node* temp_node = graph[explored_index].parent;
    // while (!temp_node->start){
       
    //     temp_node->path = true;
    //     temp_node = temp_node->parent;
    // }
    path_found_gpu = true;
  }

  if (!path_found_gpu){
    for (int i=0; i<4; i++)
    {   
      
      int new_index = explored_index + neighbor_gpu[i];
      bool edge_detect = true;

      
                
      if ((explored_index%n ==0 && neighbor_gpu[i] == -1) || (explored_index%(n-1) ==0 && neighbor_gpu[i] == 1 &&explored_index!=0) || new_index<0 || new_index >= n*n){
        edge_detect = false;
      }


      if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
      {
        graph[new_index].g = graph[explored_index].g + 1;
          
        float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2));
          // printf("%f", h_1);
        graph[new_index].h = h_1;

          
        graph[new_index].f = graph[new_index].h + graph[new_index].g;
        graph[new_index].parent = explored_index;
        graph[new_index].frontier = true;
        
        new_q[4*tid+i] = new_index;
      }
      else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
      {
        if (graph[new_index].g > graph[explored_index].g + 1)
        {
          graph[new_index].g = graph[explored_index].g + 1;
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
        }
      }
    }

  }

}
extern "C"
void parallel_explore(planner::Node* graph, int n, int start_index, int goal_index, int max_thread, std::vector<int>& path_to_goal){

  //Setup everything for planning
  bool path_found = false;
  int goal = goal_index;
  thrust::host_vector<int> q_lists;
  q_lists.push_back(start_index);

  const int map_size = n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  int neighbor[4] = {1, -1, n, -n};

  //Copy all needed variables to gpu
  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  4*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));

  //q list on gpu
  thrust::device_vector<int> q_lists_gpu = q_lists;

  while(q_lists_gpu.size()!=0 && !path_found){
    int q_size = q_lists_gpu.size();

    //new_q is the list store the frontier generated from this step of exploration
    thrust::device_vector<int> new_q_lists_gpu(4*q_size);
    thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);


    //Determine how many thread should be launched
    int thread_size = min(max_thread, q_size);


    //Launch the kernel to explore the map
    explore<<<1,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()));
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );
    // hipMemcpy(&graph, map_gpu,  map_size, hipMemcpyDeviceToHost );


    // Remove all element that is not used during the exploration and repeated value
    
    new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
    
    new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );
    
    // Create new q list based on origional and updated q
    if (q_size <= max_thread) {
      q_lists_gpu.clear();
      q_lists_gpu = new_q_lists_gpu;
      new_q_lists_gpu.clear();
    }
    else {
      
      q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+max_thread );
      q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
      new_q_lists_gpu.clear();

      // //sort the q_list based on the f value
      thrust::device_vector<float> f_value(q_lists_gpu.size());
      get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
      thrust::sort_by_key(f_value.begin(), f_value.end(), q_lists_gpu.begin() );
    }

    
    //q_size = q_lists_gpu.size();
    // thrust::device_vector<float> h_value(q_size);

    // if (q_size > 1024) {
    //   int block = q_size / 1024 + 1;
      
    //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

    //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

    // }

    if (path_found){
      hipMemcpy(graph, map_gpu,  map_size, hipMemcpyDeviceToHost );
      int path1 = goal;
      while (path1 != start_index)
        {  
          path_to_goal.push_back(path1);
          graph[path1].path = true;
          // path.push_back(path1);
          path1 = graph[path1].parent;
        }


    }
  }


 
}
