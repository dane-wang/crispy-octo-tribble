#include "hip/hip_runtime.h"
#include <ros/ros.h> 
//#include <geometry_msgs>
// #include "geometry_msgs/Point.h"
#include <time.h>  
#include <vector>
#include "graph_search/my_msg.h"
#include "graph_search/planner1.h"
#include <algorithm>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>

__device__ bool path_found_gpu;
__device__ int neighbor_gpu[4];
__device__ int goal_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T> 
__global__ void get_h(T* q,  planner::Node* graph, float* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].h;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q  )
{
  int tid = threadIdx.x;
  int explored_index = q[tid];
  int n = neighbor_gpu[3];

  graph[explored_index].explored = true;
  graph[explored_index].frontier = false;

  if (graph[explored_index].goal){
    printf("FOUND");
    printf("Hello from thread %d, I am explored %d\n", tid, explored_index);
    // planner::Node* temp_node = graph[explored_index].parent;
    // while (!temp_node->start){
       
    //     temp_node->path = true;
    //     temp_node = temp_node->parent;
    // }
    path_found_gpu = true;
  }

  if (!path_found_gpu){
    for (int i=0; i<4; i++)
    {   

      int new_index = explored_index + neighbor_gpu[i];
      bool edge_detect = true;

      if (new_index >=0) {
                
        if ((explored_index%n ==0 && neighbor_gpu[i] == -1) || (explored_index%(n-1) ==0 && neighbor_gpu[i] == 1) || new_index<0 || new_index >= n*n){
          edge_detect = false;
        }


        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + 1;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2));
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[4*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + 1)
          {
            graph[new_index].g = graph[explored_index].g + 1;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }

      }
        

    }

  }

}



  


int main(int argc, char** argv)
{
  ros::init(argc, argv, "parallel_planning");

  int n = 100;

  // planner::Node* graph1;
  // graph1 = (planner::Node*)malloc(sizeof(planner::Node)*2000*2000);
  planner::Node graph[n*n];

  

  // map initialization
  for (int y =0; y<n; y++){

    for (int x=0; x<n; x++){

      graph[y*n+x].x = x;
      graph[y*n+x].y = y;
    }
  }

  // thrust::host_vector<int> H(4);

  // std::cout << "a " << H.size() << std::endl;

  // Initialize the start and goal node
  
  int start = 10;
  int goal = 7000;

  int path1 = goal;
  bool path_found = false;

  graph[start].start = true;
  graph[start].g = 0;
  graph[start].h = planner::h_calculation(&graph[start], &graph[goal]);
  graph[start].f = graph[start].h + graph[start].g;
  graph[start].explored = true;

  graph[goal].goal = true;
  graph[goal].h = 0;


  graph[350].obstacle = true;
  graph[341].obstacle = true;
  graph[320].obstacle = true;
  graph[71].obstacle = true;
  int neighbor[4] = {1, -1, n, -n};

  // Create the priority queue for frontier
  

  // Start to work with CUDA

  thrust::host_vector<int> q_lists;

  q_lists.push_back(start);

  // Start to allocate memory on gpu:
  
  const int map_size = n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  


  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, &graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  4*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));


  thrust::device_vector<int> q_lists_gpu = q_lists;

  ros::NodeHandle nh; 

    // 发布消息 话题名字 队列大小
	ros::Publisher pub = nh.advertise<graph_search::my_msg> ("start_and_goal", 100);
    
    //geometry_msgs::Point start_goal;
  graph_search::my_msg map;


  while (ros::ok()) {
    while(ros::ok() && q_lists_gpu.size()!=0 && !path_found){

      int q_size = q_lists_gpu.size();
      thrust::device_vector<int> new_q_lists_gpu(4*q_size);
      thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);

      int thread_size = min(1024, q_size);

 
      
      explore<<<1,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()));
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );

      hipMemcpy(&graph, map_gpu,  map_size, hipMemcpyDeviceToHost );

      // Remove all element that is not used during the exploration and repeated value
      
      new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
      new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );

      std::cout << new_q_lists_gpu.size() << std::endl;

      // for (int o =0; o< new_q_lists_gpu.size(); o++){
      //   int as = new_q_lists_gpu[o];

        
      //   // printf("%d\n", as);
        
      // }

      // for (int o =0; o< new_q_lists_gpu.size(); o++){
      //   int ss = new_q_lists_gpu[0];
      //   printf("%d\n", ss);
      // }
      
      // Create new q list based on origional and updated q
      if (q_size <= 1024) {
        q_lists_gpu.clear();
        q_lists_gpu = new_q_lists_gpu;
        new_q_lists_gpu.clear();
      }
      else {
        
        q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+1024 );
        q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
      }

      
      //q_size = q_lists_gpu.size();
      // thrust::device_vector<float> h_value(q_size);

      // if (q_size > 1024) {
      //   int block = q_size / 1024 + 1;
        
      //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

      //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

      // }

      if (path_found){
        while (path1 != start)
          {
              graph[path1].path = true;
              path1 = graph[path1].parent;
          }


      }

      

      std::vector<int8_t> v(n*n, 0);
      for (int y =0; y<n; y++)
      { 
        for (int x=0; x<n; x++){

          if (graph[y*n+x].start) {
         
            v[y*n+x] = 120;
          }
          else if (graph[y*n+x].goal)
          {
            v[y*n+x] = 140;
          }
          else if (graph[y*n+x].path){
            v[y*n+x] = 250;
          }
          else if (graph[y*n+x].obstacle){
            v[y*n+x] = 100;
          }
          else if (graph[y*n+x].frontier){
            v[y*n+x] = 50;
          }
          else if (graph[y*n+x].explored){
            v[y*n+x] = 200;
          }
            
            
        }
      }
      // for (int k =0; k< n*n; k++){

      //   std::cout<< static_cast<int16_t>(v[k]) << std::endl;

      // }
     

      

      ros::Rate loop_rate(5);
            
      map.points = v;

      // map.points[10] = 120;
      // map.points[125] = 140;
  
  
      // 广播
      pub.publish(map);
      ros::spinOnce(); 
      loop_rate.sleep(); 

    }
    if (path_found){
      ros::Rate loop_rate(5);
      pub.publish(map);
      ros::spinOnce(); 
      loop_rate.sleep(); 

  }

  }


  


  return 0;
}
