#include "hip/hip_runtime.h"

//#include <geometry_msgs>
// #include "geometry_msgs/Point.h"
#include <time.h>  
#include <vector>
#include <chrono>
#include "planner1.h"
#include <algorithm>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>

__device__ bool path_found_gpu;
__device__ int neighbor_gpu[4];
__device__ int goal_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T> 
__global__ void get_h(T* q,  planner::Node* graph, float* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].h;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q  )
{
  int tid = threadIdx.x;
  int explored_index = q[tid];
  int n = neighbor_gpu[3];

  graph[explored_index].explored = true;
  graph[explored_index].frontier = false;

  if (graph[explored_index].goal){
    printf("FOUND");
    printf("Hello from thread %d, I am explored %d\n", tid, explored_index);
    // planner::Node* temp_node = graph[explored_index].parent;
    // while (!temp_node->start){
       
    //     temp_node->path = true;
    //     temp_node = temp_node->parent;
    // }
    path_found_gpu = true;
  }

  if (!path_found_gpu){
    for (int i=0; i<4; i++)
    {   

      int new_index = explored_index + neighbor_gpu[i];
      bool edge_detect = true;

      if (new_index >=0) {
                
        if ((explored_index%n ==0 && neighbor_gpu[i] == -1) || (explored_index%(n-1) ==0 && neighbor_gpu[i] == 1) || new_index<0 || new_index >= n*n){
          edge_detect = false;
        }


        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + 1;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2));
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[4*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + 1)
          {
            graph[new_index].g = graph[explored_index].g + 1;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }

      }
        

    }

  }

}



  


int main(int argc, char** argv)
{
  
  using std::chrono::high_resolution_clock;
  using std::chrono::duration_cast;
  using std::chrono::duration;
  using std::chrono::milliseconds;
	auto t1 = high_resolution_clock::now();
  int n = 120;

  // planner::Node* graph1;
  // graph1 = (planner::Node*)malloc(sizeof(planner::Node)*2000*2000);
  planner::Node graph[n*n];

  

  // map initialization
  for (int y =0; y<n; y++){

    for (int x=0; x<n; x++){

      graph[y*n+x].x = x;
      graph[y*n+x].y = y;
    }
  }

  // thrust::host_vector<int> H(4);

  // std::cout << "a " << H.size() << std::endl;

  // Initialize the start and goal node
  
  int start = 1;
  int goal = n*n-1;

  int path1 = goal;
  bool path_found = false;

  graph[start].start = true;
  graph[start].g = 0;
  graph[start].h = planner::h_calculation(&graph[start], &graph[goal]);
  graph[start].f = graph[start].h + graph[start].g;
  graph[start].explored = true;

  graph[goal].goal = true;
  graph[goal].h = 0;


  graph[350].obstacle = true;
  graph[341].obstacle = true;
  graph[320].obstacle = true;
  graph[71].obstacle = true;
  int neighbor[4] = {1, -1, n, -n};

  // Create the priority queue for frontier
  

  // Start to work with CUDA

  thrust::host_vector<int> q_lists;

  q_lists.push_back(start);

  // Start to allocate memory on gpu:
  
  const int map_size = n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  


  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, &graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  4*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));

  auto t3 = high_resolution_clock::now();
  duration<double, std::milli> ms_double = t3 - t1;
  std::cout <<"Initialization is \t" << ms_double.count() << "ms\n";


  thrust::device_vector<int> q_lists_gpu = q_lists;

  
  while(q_lists_gpu.size()!=0 && !path_found){

    int q_size = q_lists_gpu.size();
    thrust::device_vector<int> new_q_lists_gpu(4*q_size);
    thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);

    int thread_size = min(1024, q_size);


    
    explore<<<1,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()));
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );

  

    // Remove all element that is not used during the exploration and repeated value
    
    new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
    new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );

    // std::cout << new_q_lists_gpu.size() << std::endl;

    // for (int o =0; o< new_q_lists_gpu.size(); o++){
    //   int as = new_q_lists_gpu[o];

      
    //   // printf("%d\n", as);
      
    // }

    // for (int o =0; o< new_q_lists_gpu.size(); o++){
    //   int ss = new_q_lists_gpu[0];
    //   printf("%d\n", ss);
    // }
    
    // Create new q list based on origional and updated q
    if (q_size <= 1024) {
      q_lists_gpu.clear();
      q_lists_gpu = new_q_lists_gpu;
      new_q_lists_gpu.clear();
    }
    else {
      
      q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+1024 );
      q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
    }

    
    //q_size = q_lists_gpu.size();
    // thrust::device_vector<float> h_value(q_size);

    // if (q_size > 1024) {
    //   int block = q_size / 1024 + 1;
      
    //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

    //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

    // }

    if (path_found){
      hipMemcpy(&graph, map_gpu,  map_size, hipMemcpyDeviceToHost );
      auto t2 = high_resolution_clock::now();
      duration<double, std::milli> ms_double = t2 - t1;
      std::cout <<"Total time is \t" << ms_double.count() << "ms\n";
      while (path1 != start)
        {
            graph[path1].path = true;
            path1 = graph[path1].parent;
        }


      }

      

      // for (int k =0; k< n*n; k++){


    
   

  }

  


  


  return 0;
}
